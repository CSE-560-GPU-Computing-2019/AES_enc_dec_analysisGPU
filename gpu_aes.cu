//GPU AES Encryption and Decryption
/*
Aamir Tufail Ahmad - 2016001
Arnav Kumar - 2016017
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h> 
#include <string.h>
#include <math.h>

__device__ void GPU_byteSubShiftRow(unsigned char * d_state, unsigned char * s_s)
{

    unsigned char tmp[16];

    tmp[0] = s_s[d_state[0]];    tmp[1] = s_s[d_state[5]];    tmp[2] = s_s[d_state[10]];    tmp[3] = s_s[d_state[15]];

    tmp[4] = s_s[d_state[4]];    tmp[5] = s_s[d_state[9]];    tmp[6] = s_s[d_state[14]];    tmp[7] = s_s[d_state[3]];

    tmp[8] = s_s[d_state[8]];    tmp[9] = s_s[d_state[13]];   tmp[10] = s_s[d_state[2]];    tmp[11] = s_s[d_state[7]];

    tmp[12] = s_s[d_state[12]];  tmp[13] = s_s[d_state[1]];   tmp[14] = s_s[d_state[6]];    tmp[15] = s_s[d_state[11]];


    for(int i=0;i<16;i++)
    {
        d_state[i] = tmp[i];
    }
}

__device__ void GPU_mixColumns(unsigned char * d_plainText, unsigned char * s_mul2, unsigned char * s_mul_3){
    unsigned char d_tempC[16];

    for (int i = 0; i < 4; ++i){
        d_tempC[(4*i)+0] = (unsigned char) (s_mul2[ d_plainText[(4*i)+0]] ^ s_mul_3[ d_plainText[(4*i)+1]] ^ d_plainText[(4*i)+2] ^ d_plainText[(4*i)+3]);
        d_tempC[(4*i)+1] = (unsigned char) (d_plainText[(4*i)+0] ^ s_mul2[ d_plainText[(4*i)+1]] ^ s_mul_3[ d_plainText[(4*i)+2]] ^ d_plainText[(4*i)+3]);
        d_tempC[(4*i)+2] = (unsigned char) (d_plainText[(4*i)+0] ^ d_plainText[(4*i)+1] ^ s_mul2[ d_plainText[(4*i)+2]] ^ s_mul_3[ d_plainText[(4*i)+3]]);
        d_tempC[(4*i)+3] = (unsigned char) (s_mul_3[ d_plainText[(4*i)+0]] ^ d_plainText[(4*i)+1] ^ d_plainText[(4*i)+2] ^ s_mul2[ d_plainText[(4*i)+3]]);
    }

    for (int i = 0; i < 16; ++i){
        d_plainText[i] = d_tempC[i];
    }
}


__global__ void GPU_AESEncryption(unsigned char * d_plainText, unsigned char * d_expandedKey, unsigned char * d_cipher, 
    unsigned char * d_s, unsigned char * d_mul2, unsigned char * d_mul_3, int len){

    unsigned char  d_state [16];

    __shared__ unsigned char s_expandedKey[256],s_s[256],s_mul2[256],s_mul_3[256];

    s_s[threadIdx.x]=d_s[threadIdx.x];
    s_mul2[threadIdx.x]=d_mul2[threadIdx.x];
    s_mul_3[threadIdx.x]=d_mul_3[threadIdx.x];
    s_expandedKey[threadIdx.x]=d_expandedKey[threadIdx.x];



    int pos=(threadIdx.x+blockIdx.x*blockDim.x)*16;
    if(pos>len)
        return;

    for (int i = 0; i < 16; ++i){
     d_state[i] = d_plainText[pos+i] ^ s_expandedKey[i];
    }

    for(int rounds = 1; rounds<10; rounds++)
    {
        GPU_byteSubShiftRow(d_state,s_s);
        GPU_mixColumns(d_state,s_mul2,s_mul_3);
        int counter = 0;
        int loc = rounds*16;
        while(counter<16){
            d_state[counter] ^= s_expandedKey[loc];
            loc++;
            counter++;
        }
    }

    //10th round
    GPU_byteSubShiftRow(d_state,s_s);
    for(int i=0; i<16;i++){
        d_cipher[pos+i] = d_state[i] ^ s_expandedKey[i+160];
    }
    
}
__device__ void GPU_inverseByteSubShiftRow(unsigned char * d_plainText, unsigned char * s_inv_s)
{
    unsigned char temp[16];
    temp[0] = s_inv_s[d_plainText[0]];
    temp[1] = s_inv_s[d_plainText[13]];
    temp[2] = s_inv_s[d_plainText[10]];
    temp[3] = s_inv_s[d_plainText[7]];
    temp[4] = s_inv_s[d_plainText[4]];
    temp[5] = s_inv_s[d_plainText[1]];
    temp[6] = s_inv_s[d_plainText[14]];
    temp[7] = s_inv_s[d_plainText[11]];
    temp[8] = s_inv_s[d_plainText[8]];
    temp[9] = s_inv_s[d_plainText[5]];
    temp[10] = s_inv_s[d_plainText[2]];
    temp[11] = s_inv_s[d_plainText[15]];
    temp[12] = s_inv_s[d_plainText[12]];
    temp[13] = s_inv_s[d_plainText[9]];
    temp[14] = s_inv_s[d_plainText[6]];
    temp[15] = s_inv_s[d_plainText[3]];

    for (int i = 0; i < 16; ++i)
        d_plainText[i] = temp[i];
}

__device__ void GPU_inverseMixedColumn(unsigned char * d_plainText, unsigned char * s_mul_14, unsigned char * s_mul_9,unsigned char * s_mul_13,unsigned char * s_mul_11){

    unsigned char tempC [18];

    for (int i = 0; i < 4; ++i)
    {
        tempC[(4*i)+0] = (unsigned char) (s_mul_14[d_plainText[(4*i)+0]] ^ s_mul_11[d_plainText[(4*i)+1]] ^ s_mul_13[d_plainText[(4*i)+2]] ^ s_mul_9[d_plainText[(4*i)+3]]);
        tempC[(4*i)+1] = (unsigned char) (s_mul_9[d_plainText[(4*i)+0]] ^ s_mul_14[d_plainText[(4*i)+1]] ^ s_mul_11[d_plainText[(4*i)+2]] ^ s_mul_13[d_plainText[(4*i)+3]]);
        tempC[(4*i)+2] = (unsigned char) (s_mul_13[d_plainText[(4*i)+0]] ^ s_mul_9[d_plainText[(4*i)+1]] ^ s_mul_14[d_plainText[(4*i)+2]] ^ s_mul_11[d_plainText[(4*i)+3]]);
        tempC[(4*i)+3] = (unsigned char) (s_mul_11[d_plainText[(4*i)+0]] ^ s_mul_13[d_plainText[(4*i)+1]] ^ s_mul_9[d_plainText[(4*i)+2]] ^ s_mul_14[d_plainText[(4*i)+3]]);
    }
    for (int i = 0; i < 16; ++i)
    {
        d_plainText[i] = tempC[i];
    }

}

__global__ void GPU_AESDecryption(unsigned char * d_plainText, unsigned char * d_expandedKey, unsigned char * d_cipher, 
    unsigned char * d_inv_s, unsigned char * d_mul_14, unsigned char * d_mul_9, unsigned char * d_mul_13, unsigned char * d_mul_11, int len){


        unsigned char  d_state [16];

        __shared__ unsigned char s_expandedKey[256],s_s_inv[256],s_mul_14[256],s_mul_9[256],s_mul_13[256],s_mul_11[256];

        s_s_inv[threadIdx.x]=d_inv_s[threadIdx.x];
        s_mul_14[threadIdx.x]=d_mul_14[threadIdx.x];
        s_mul_9[threadIdx.x]=d_mul_9[threadIdx.x];
        s_mul_13[threadIdx.x]=d_mul_13[threadIdx.x];
        s_mul_11[threadIdx.x]=d_mul_11[threadIdx.x];
        s_expandedKey[threadIdx.x]=d_expandedKey[threadIdx.x];

        int pos=(threadIdx.x+blockIdx.x*blockDim.x)*16;
        if(pos>len)
            return;

        //key whitening
        for (int i = 0; i < 16; ++i)
            d_state[i] = d_cipher[i] ^ s_expandedKey[160+i];

        // 9 rounds of decryption
        for (int rounds = 9; rounds >0 ; rounds--)
        {
            GPU_inverseByteSubShiftRow(d_state,s_s_inv);
            int counter = 0;
            int loc = 16*rounds;
            while(counter<16)
            {
                d_state[counter] ^= s_expandedKey[loc];
                loc++;
                counter++;
            }
            GPU_inverseMixedColumn(d_state,s_mul_14,s_mul_9,s_mul_13,s_mul_11);
        }

        //final 10th round of decryption
        GPU_inverseByteSubShiftRow(d_state,s_s_inv);
        for(int i =0; i<16; i++)
            d_plainText[i] = d_state[i] ^ s_expandedKey[i];
    }

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

unsigned char s[256] = 
 {
    0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
    0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
    0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
    0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
    0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
    0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
    0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
    0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
    0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
    0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
    0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
    0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
    0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
    0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
    0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
    0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
 };

 unsigned char inv_s[256] = 
 {
    0x52, 0x09, 0x6A, 0xD5, 0x30, 0x36, 0xA5, 0x38, 0xBF, 0x40, 0xA3, 0x9E, 0x81, 0xF3, 0xD7, 0xFB,
    0x7C, 0xE3, 0x39, 0x82, 0x9B, 0x2F, 0xFF, 0x87, 0x34, 0x8E, 0x43, 0x44, 0xC4, 0xDE, 0xE9, 0xCB,
    0x54, 0x7B, 0x94, 0x32, 0xA6, 0xC2, 0x23, 0x3D, 0xEE, 0x4C, 0x95, 0x0B, 0x42, 0xFA, 0xC3, 0x4E,
    0x08, 0x2E, 0xA1, 0x66, 0x28, 0xD9, 0x24, 0xB2, 0x76, 0x5B, 0xA2, 0x49, 0x6D, 0x8B, 0xD1, 0x25,
    0x72, 0xF8, 0xF6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xD4, 0xA4, 0x5C, 0xCC, 0x5D, 0x65, 0xB6, 0x92,
    0x6C, 0x70, 0x48, 0x50, 0xFD, 0xED, 0xB9, 0xDA, 0x5E, 0x15, 0x46, 0x57, 0xA7, 0x8D, 0x9D, 0x84,
    0x90, 0xD8, 0xAB, 0x00, 0x8C, 0xBC, 0xD3, 0x0A, 0xF7, 0xE4, 0x58, 0x05, 0xB8, 0xB3, 0x45, 0x06,
    0xD0, 0x2C, 0x1E, 0x8F, 0xCA, 0x3F, 0x0F, 0x02, 0xC1, 0xAF, 0xBD, 0x03, 0x01, 0x13, 0x8A, 0x6B,
    0x3A, 0x91, 0x11, 0x41, 0x4F, 0x67, 0xDC, 0xEA, 0x97, 0xF2, 0xCF, 0xCE, 0xF0, 0xB4, 0xE6, 0x73,
    0x96, 0xAC, 0x74, 0x22, 0xE7, 0xAD, 0x35, 0x85, 0xE2, 0xF9, 0x37, 0xE8, 0x1C, 0x75, 0xDF, 0x6E,
    0x47, 0xF1, 0x1A, 0x71, 0x1D, 0x29, 0xC5, 0x89, 0x6F, 0xB7, 0x62, 0x0E, 0xAA, 0x18, 0xBE, 0x1B,
    0xFC, 0x56, 0x3E, 0x4B, 0xC6, 0xD2, 0x79, 0x20, 0x9A, 0xDB, 0xC0, 0xFE, 0x78, 0xCD, 0x5A, 0xF4,
    0x1F, 0xDD, 0xA8, 0x33, 0x88, 0x07, 0xC7, 0x31, 0xB1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xEC, 0x5F,
    0x60, 0x51, 0x7F, 0xA9, 0x19, 0xB5, 0x4A, 0x0D, 0x2D, 0xE5, 0x7A, 0x9F, 0x93, 0xC9, 0x9C, 0xEF,
    0xA0, 0xE0, 0x3B, 0x4D, 0xAE, 0x2A, 0xF5, 0xB0, 0xC8, 0xEB, 0xBB, 0x3C, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2B, 0x04, 0x7E, 0xBA, 0x77, 0xD6, 0x26, 0xE1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0C, 0x7D
 };

unsigned char mul2[] =
{
    0x00,0x02,0x04,0x06,0x08,0x0a,0x0c,0x0e,0x10,0x12,0x14,0x16,0x18,0x1a,0x1c,0x1e,
    0x20,0x22,0x24,0x26,0x28,0x2a,0x2c,0x2e,0x30,0x32,0x34,0x36,0x38,0x3a,0x3c,0x3e,
    0x40,0x42,0x44,0x46,0x48,0x4a,0x4c,0x4e,0x50,0x52,0x54,0x56,0x58,0x5a,0x5c,0x5e,
    0x60,0x62,0x64,0x66,0x68,0x6a,0x6c,0x6e,0x70,0x72,0x74,0x76,0x78,0x7a,0x7c,0x7e,
    0x80,0x82,0x84,0x86,0x88,0x8a,0x8c,0x8e,0x90,0x92,0x94,0x96,0x98,0x9a,0x9c,0x9e,
    0xa0,0xa2,0xa4,0xa6,0xa8,0xaa,0xac,0xae,0xb0,0xb2,0xb4,0xb6,0xb8,0xba,0xbc,0xbe,
    0xc0,0xc2,0xc4,0xc6,0xc8,0xca,0xcc,0xce,0xd0,0xd2,0xd4,0xd6,0xd8,0xda,0xdc,0xde,
    0xe0,0xe2,0xe4,0xe6,0xe8,0xea,0xec,0xee,0xf0,0xf2,0xf4,0xf6,0xf8,0xfa,0xfc,0xfe,
    0x1b,0x19,0x1f,0x1d,0x13,0x11,0x17,0x15,0x0b,0x09,0x0f,0x0d,0x03,0x01,0x07,0x05,
    0x3b,0x39,0x3f,0x3d,0x33,0x31,0x37,0x35,0x2b,0x29,0x2f,0x2d,0x23,0x21,0x27,0x25,
    0x5b,0x59,0x5f,0x5d,0x53,0x51,0x57,0x55,0x4b,0x49,0x4f,0x4d,0x43,0x41,0x47,0x45,
    0x7b,0x79,0x7f,0x7d,0x73,0x71,0x77,0x75,0x6b,0x69,0x6f,0x6d,0x63,0x61,0x67,0x65,
    0x9b,0x99,0x9f,0x9d,0x93,0x91,0x97,0x95,0x8b,0x89,0x8f,0x8d,0x83,0x81,0x87,0x85,
    0xbb,0xb9,0xbf,0xbd,0xb3,0xb1,0xb7,0xb5,0xab,0xa9,0xaf,0xad,0xa3,0xa1,0xa7,0xa5,
    0xdb,0xd9,0xdf,0xdd,0xd3,0xd1,0xd7,0xd5,0xcb,0xc9,0xcf,0xcd,0xc3,0xc1,0xc7,0xc5,
    0xfb,0xf9,0xff,0xfd,0xf3,0xf1,0xf7,0xf5,0xeb,0xe9,0xef,0xed,0xe3,0xe1,0xe7,0xe5
};

unsigned char mul_3[] = 
{ 
    0x00,0x03,0x06,0x05,0x0c,0x0f,0x0a,0x09,0x18,0x1b,0x1e,0x1d,0x14,0x17,0x12,0x11,
    0x30,0x33,0x36,0x35,0x3c,0x3f,0x3a,0x39,0x28,0x2b,0x2e,0x2d,0x24,0x27,0x22,0x21,
    0x60,0x63,0x66,0x65,0x6c,0x6f,0x6a,0x69,0x78,0x7b,0x7e,0x7d,0x74,0x77,0x72,0x71,
    0x50,0x53,0x56,0x55,0x5c,0x5f,0x5a,0x59,0x48,0x4b,0x4e,0x4d,0x44,0x47,0x42,0x41,
    0xc0,0xc3,0xc6,0xc5,0xcc,0xcf,0xca,0xc9,0xd8,0xdb,0xde,0xdd,0xd4,0xd7,0xd2,0xd1,
    0xf0,0xf3,0xf6,0xf5,0xfc,0xff,0xfa,0xf9,0xe8,0xeb,0xee,0xed,0xe4,0xe7,0xe2,0xe1,
    0xa0,0xa3,0xa6,0xa5,0xac,0xaf,0xaa,0xa9,0xb8,0xbb,0xbe,0xbd,0xb4,0xb7,0xb2,0xb1,
    0x90,0x93,0x96,0x95,0x9c,0x9f,0x9a,0x99,0x88,0x8b,0x8e,0x8d,0x84,0x87,0x82,0x81,
    0x9b,0x98,0x9d,0x9e,0x97,0x94,0x91,0x92,0x83,0x80,0x85,0x86,0x8f,0x8c,0x89,0x8a,
    0xab,0xa8,0xad,0xae,0xa7,0xa4,0xa1,0xa2,0xb3,0xb0,0xb5,0xb6,0xbf,0xbc,0xb9,0xba,
    0xfb,0xf8,0xfd,0xfe,0xf7,0xf4,0xf1,0xf2,0xe3,0xe0,0xe5,0xe6,0xef,0xec,0xe9,0xea,
    0xcb,0xc8,0xcd,0xce,0xc7,0xc4,0xc1,0xc2,0xd3,0xd0,0xd5,0xd6,0xdf,0xdc,0xd9,0xda,
    0x5b,0x58,0x5d,0x5e,0x57,0x54,0x51,0x52,0x43,0x40,0x45,0x46,0x4f,0x4c,0x49,0x4a,
    0x6b,0x68,0x6d,0x6e,0x67,0x64,0x61,0x62,0x73,0x70,0x75,0x76,0x7f,0x7c,0x79,0x7a,
    0x3b,0x38,0x3d,0x3e,0x37,0x34,0x31,0x32,0x23,0x20,0x25,0x26,0x2f,0x2c,0x29,0x2a,
    0x0b,0x08,0x0d,0x0e,0x07,0x04,0x01,0x02,0x13,0x10,0x15,0x16,0x1f,0x1c,0x19,0x1a
};

unsigned char mul_9[] = 
{
    0x00,0x09,0x12,0x1b,0x24,0x2d,0x36,0x3f,0x48,0x41,0x5a,0x53,0x6c,0x65,0x7e,0x77,
    0x90,0x99,0x82,0x8b,0xb4,0xbd,0xa6,0xaf,0xd8,0xd1,0xca,0xc3,0xfc,0xf5,0xee,0xe7,
    0x3b,0x32,0x29,0x20,0x1f,0x16,0x0d,0x04,0x73,0x7a,0x61,0x68,0x57,0x5e,0x45,0x4c,
    0xab,0xa2,0xb9,0xb0,0x8f,0x86,0x9d,0x94,0xe3,0xea,0xf1,0xf8,0xc7,0xce,0xd5,0xdc,
    0x76,0x7f,0x64,0x6d,0x52,0x5b,0x40,0x49,0x3e,0x37,0x2c,0x25,0x1a,0x13,0x08,0x01,
    0xe6,0xef,0xf4,0xfd,0xc2,0xcb,0xd0,0xd9,0xae,0xa7,0xbc,0xb5,0x8a,0x83,0x98,0x91,
    0x4d,0x44,0x5f,0x56,0x69,0x60,0x7b,0x72,0x05,0x0c,0x17,0x1e,0x21,0x28,0x33,0x3a,
    0xdd,0xd4,0xcf,0xc6,0xf9,0xf0,0xeb,0xe2,0x95,0x9c,0x87,0x8e,0xb1,0xb8,0xa3,0xaa,
    0xec,0xe5,0xfe,0xf7,0xc8,0xc1,0xda,0xd3,0xa4,0xad,0xb6,0xbf,0x80,0x89,0x92,0x9b,
    0x7c,0x75,0x6e,0x67,0x58,0x51,0x4a,0x43,0x34,0x3d,0x26,0x2f,0x10,0x19,0x02,0x0b,
    0xd7,0xde,0xc5,0xcc,0xf3,0xfa,0xe1,0xe8,0x9f,0x96,0x8d,0x84,0xbb,0xb2,0xa9,0xa0,
    0x47,0x4e,0x55,0x5c,0x63,0x6a,0x71,0x78,0x0f,0x06,0x1d,0x14,0x2b,0x22,0x39,0x30,
    0x9a,0x93,0x88,0x81,0xbe,0xb7,0xac,0xa5,0xd2,0xdb,0xc0,0xc9,0xf6,0xff,0xe4,0xed,
    0x0a,0x03,0x18,0x11,0x2e,0x27,0x3c,0x35,0x42,0x4b,0x50,0x59,0x66,0x6f,0x74,0x7d,
    0xa1,0xa8,0xb3,0xba,0x85,0x8c,0x97,0x9e,0xe9,0xe0,0xfb,0xf2,0xcd,0xc4,0xdf,0xd6,
    0x31,0x38,0x23,0x2a,0x15,0x1c,0x07,0x0e,0x79,0x70,0x6b,0x62,0x5d,0x54,0x4f,0x46
};

unsigned char mul_11[] = 
{
    0x00,0x0b,0x16,0x1d,0x2c,0x27,0x3a,0x31,0x58,0x53,0x4e,0x45,0x74,0x7f,0x62,0x69,
    0xb0,0xbb,0xa6,0xad,0x9c,0x97,0x8a,0x81,0xe8,0xe3,0xfe,0xf5,0xc4,0xcf,0xd2,0xd9,
    0x7b,0x70,0x6d,0x66,0x57,0x5c,0x41,0x4a,0x23,0x28,0x35,0x3e,0x0f,0x04,0x19,0x12,
    0xcb,0xc0,0xdd,0xd6,0xe7,0xec,0xf1,0xfa,0x93,0x98,0x85,0x8e,0xbf,0xb4,0xa9,0xa2,
    0xf6,0xfd,0xe0,0xeb,0xda,0xd1,0xcc,0xc7,0xae,0xa5,0xb8,0xb3,0x82,0x89,0x94,0x9f,
    0x46,0x4d,0x50,0x5b,0x6a,0x61,0x7c,0x77,0x1e,0x15,0x08,0x03,0x32,0x39,0x24,0x2f,
    0x8d,0x86,0x9b,0x90,0xa1,0xaa,0xb7,0xbc,0xd5,0xde,0xc3,0xc8,0xf9,0xf2,0xef,0xe4,
    0x3d,0x36,0x2b,0x20,0x11,0x1a,0x07,0x0c,0x65,0x6e,0x73,0x78,0x49,0x42,0x5f,0x54,
    0xf7,0xfc,0xe1,0xea,0xdb,0xd0,0xcd,0xc6,0xaf,0xa4,0xb9,0xb2,0x83,0x88,0x95,0x9e,
    0x47,0x4c,0x51,0x5a,0x6b,0x60,0x7d,0x76,0x1f,0x14,0x09,0x02,0x33,0x38,0x25,0x2e,
    0x8c,0x87,0x9a,0x91,0xa0,0xab,0xb6,0xbd,0xd4,0xdf,0xc2,0xc9,0xf8,0xf3,0xee,0xe5,
    0x3c,0x37,0x2a,0x21,0x10,0x1b,0x06,0x0d,0x64,0x6f,0x72,0x79,0x48,0x43,0x5e,0x55,
    0x01,0x0a,0x17,0x1c,0x2d,0x26,0x3b,0x30,0x59,0x52,0x4f,0x44,0x75,0x7e,0x63,0x68,
    0xb1,0xba,0xa7,0xac,0x9d,0x96,0x8b,0x80,0xe9,0xe2,0xff,0xf4,0xc5,0xce,0xd3,0xd8,
    0x7a,0x71,0x6c,0x67,0x56,0x5d,0x40,0x4b,0x22,0x29,0x34,0x3f,0x0e,0x05,0x18,0x13,
    0xca,0xc1,0xdc,0xd7,0xe6,0xed,0xf0,0xfb,0x92,0x99,0x84,0x8f,0xbe,0xb5,0xa8,0xa3
};

unsigned char mul_13[] = 
{
    0x00,0x0d,0x1a,0x17,0x34,0x39,0x2e,0x23,0x68,0x65,0x72,0x7f,0x5c,0x51,0x46,0x4b,
    0xd0,0xdd,0xca,0xc7,0xe4,0xe9,0xfe,0xf3,0xb8,0xb5,0xa2,0xaf,0x8c,0x81,0x96,0x9b,
    0xbb,0xb6,0xa1,0xac,0x8f,0x82,0x95,0x98,0xd3,0xde,0xc9,0xc4,0xe7,0xea,0xfd,0xf0,
    0x6b,0x66,0x71,0x7c,0x5f,0x52,0x45,0x48,0x03,0x0e,0x19,0x14,0x37,0x3a,0x2d,0x20,
    0x6d,0x60,0x77,0x7a,0x59,0x54,0x43,0x4e,0x05,0x08,0x1f,0x12,0x31,0x3c,0x2b,0x26,
    0xbd,0xb0,0xa7,0xaa,0x89,0x84,0x93,0x9e,0xd5,0xd8,0xcf,0xc2,0xe1,0xec,0xfb,0xf6,
    0xd6,0xdb,0xcc,0xc1,0xe2,0xef,0xf8,0xf5,0xbe,0xb3,0xa4,0xa9,0x8a,0x87,0x90,0x9d,
    0x06,0x0b,0x1c,0x11,0x32,0x3f,0x28,0x25,0x6e,0x63,0x74,0x79,0x5a,0x57,0x40,0x4d,
    0xda,0xd7,0xc0,0xcd,0xee,0xe3,0xf4,0xf9,0xb2,0xbf,0xa8,0xa5,0x86,0x8b,0x9c,0x91,
    0x0a,0x07,0x10,0x1d,0x3e,0x33,0x24,0x29,0x62,0x6f,0x78,0x75,0x56,0x5b,0x4c,0x41,
    0x61,0x6c,0x7b,0x76,0x55,0x58,0x4f,0x42,0x09,0x04,0x13,0x1e,0x3d,0x30,0x27,0x2a,
    0xb1,0xbc,0xab,0xa6,0x85,0x88,0x9f,0x92,0xd9,0xd4,0xc3,0xce,0xed,0xe0,0xf7,0xfa,
    0xb7,0xba,0xad,0xa0,0x83,0x8e,0x99,0x94,0xdf,0xd2,0xc5,0xc8,0xeb,0xe6,0xf1,0xfc,
    0x67,0x6a,0x7d,0x70,0x53,0x5e,0x49,0x44,0x0f,0x02,0x15,0x18,0x3b,0x36,0x21,0x2c,
    0x0c,0x01,0x16,0x1b,0x38,0x35,0x22,0x2f,0x64,0x69,0x7e,0x73,0x50,0x5d,0x4a,0x47,
    0xdc,0xd1,0xc6,0xcb,0xe8,0xe5,0xf2,0xff,0xb4,0xb9,0xae,0xa3,0x80,0x8d,0x9a,0x97
};

unsigned char mul_14[] = 
{
    0x00,0x0e,0x1c,0x12,0x38,0x36,0x24,0x2a,0x70,0x7e,0x6c,0x62,0x48,0x46,0x54,0x5a,
    0xe0,0xee,0xfc,0xf2,0xd8,0xd6,0xc4,0xca,0x90,0x9e,0x8c,0x82,0xa8,0xa6,0xb4,0xba,
    0xdb,0xd5,0xc7,0xc9,0xe3,0xed,0xff,0xf1,0xab,0xa5,0xb7,0xb9,0x93,0x9d,0x8f,0x81,
    0x3b,0x35,0x27,0x29,0x03,0x0d,0x1f,0x11,0x4b,0x45,0x57,0x59,0x73,0x7d,0x6f,0x61,
    0xad,0xa3,0xb1,0xbf,0x95,0x9b,0x89,0x87,0xdd,0xd3,0xc1,0xcf,0xe5,0xeb,0xf9,0xf7,
    0x4d,0x43,0x51,0x5f,0x75,0x7b,0x69,0x67,0x3d,0x33,0x21,0x2f,0x05,0x0b,0x19,0x17,
    0x76,0x78,0x6a,0x64,0x4e,0x40,0x52,0x5c,0x06,0x08,0x1a,0x14,0x3e,0x30,0x22,0x2c,
    0x96,0x98,0x8a,0x84,0xae,0xa0,0xb2,0xbc,0xe6,0xe8,0xfa,0xf4,0xde,0xd0,0xc2,0xcc,
    0x41,0x4f,0x5d,0x53,0x79,0x77,0x65,0x6b,0x31,0x3f,0x2d,0x23,0x09,0x07,0x15,0x1b,
    0xa1,0xaf,0xbd,0xb3,0x99,0x97,0x85,0x8b,0xd1,0xdf,0xcd,0xc3,0xe9,0xe7,0xf5,0xfb,
    0x9a,0x94,0x86,0x88,0xa2,0xac,0xbe,0xb0,0xea,0xe4,0xf6,0xf8,0xd2,0xdc,0xce,0xc0,
    0x7a,0x74,0x66,0x68,0x42,0x4c,0x5e,0x50,0x0a,0x04,0x16,0x18,0x32,0x3c,0x2e,0x20,
    0xec,0xe2,0xf0,0xfe,0xd4,0xda,0xc8,0xc6,0x9c,0x92,0x80,0x8e,0xa4,0xaa,0xb8,0xb6,
    0x0c,0x02,0x10,0x1e,0x34,0x3a,0x28,0x26,0x7c,0x72,0x60,0x6e,0x44,0x4a,0x58,0x56,
    0x37,0x39,0x2b,0x25,0x0f,0x01,0x13,0x1d,0x47,0x49,0x5b,0x55,0x7f,0x71,0x63,0x6d,
    0xd7,0xd9,0xcb,0xc5,0xef,0xe1,0xf3,0xfd,0xa7,0xa9,0xbb,0xb5,0x9f,0x91,0x83,0x8d
};

unsigned char rcon[11] = 
{
    0x01000000, 0x02000000, 0x04000000, 0x08000000, 0x10000000, 0x20000000,
    0x40000000, 0x80000000, 0x1b000000, 0x36000000
};

unsigned char wReady[4];
unsigned char * g (unsigned char wInput[4], int counter)
{
    
    unsigned char temp[4] = "";
    unsigned char a = wInput[0];
    for(int i =0;i<3; i++)
    {
        temp[i] = wInput[(i+1)];
    }
    temp[3] = a;

    for (int i =0; i<4;i++)
        temp[i] = s[temp[i]];

    //unsigned char array formed for xoring with rcon

    unsigned char array2[4] = "";
    array2[0] = rcon[counter];
    array2[1] = array2[2] = array2[3] = 0x00;

    for (int i=0;i<4;i++)
    wReady[i] = temp[i] ^ array2[i];
    return wReady;
}

unsigned char expandedKeyfunc[176];
unsigned char * keyExpansion(unsigned char key[16])
{

    unsigned char words[44][4];
    for (int i = 0; i < 44; ++i)
    {
        for (int j = 0; j <4; ++j)
        {
            words[i][j]=0x00;
        }
    }
    
    
    
    int byteCount = 0; //this is to keep a count on the bytes of the expandedKey array
    
    for (int i=0;i<16;i++)
            expandedKeyfunc[i] = key[i];
    // printf("expanded key : %s\n",expandedKey);

    for(int j=0;j<4;j++)
    {
         for(int k=0;k<4;k++)
         {
            words[j][k] = expandedKeyfunc[byteCount++];
            // printf("words : %s\n",words[j]);

         }
    }
    for(int l=4;l<44;l++)
    {
        if((l%4)==0)
        {
            for(int m=0;m<4;m++)
            {
                words[l][m] = words[(l-4)][m] ^ g(words[l-1], (l/4))[m];
                // printf("words : %s\n",words[l]);

                
            }
        }
        else
        {
            for(int n=0;n<4;n++)
            {
                words[l][n] = words[l-1][n] ^ words[l-4][n];
            }
        }
    }

    int loc=0;
    for(int i=0;i<44;i++ )
    {
        for(int j=0;j<4;j++)
        {
            expandedKeyfunc[loc] = words[i][j];
            loc++;
        }
    }
    // printf("expanded key : %d\n",strlen(expandedKey));
    return expandedKeyfunc;
}

void mixColumns(unsigned char * plainText)
{
    unsigned char tempC[16];

    for (int i = 0; i < 4; ++i)
    {
        tempC[(4*i)+0] = (unsigned char) (mul2[plainText[(4*i)+0]] ^ mul_3[plainText[(4*i)+1]] ^ plainText[(4*i)+2] ^ plainText[(4*i)+3]);
        tempC[(4*i)+1] = (unsigned char) (plainText[(4*i)+0] ^ mul2[plainText[(4*i)+1]] ^ mul_3[plainText[(4*i)+2]] ^ plainText[(4*i)+3]);
        tempC[(4*i)+2] = (unsigned char) (plainText[(4*i)+0] ^ plainText[(4*i)+1] ^ mul2[plainText[(4*i)+2]] ^ mul_3[plainText[(4*i)+3]]);
        tempC[(4*i)+3] = (unsigned char) (mul_3[plainText[(4*i)+0]] ^ plainText[(4*i)+1] ^ plainText[(4*i)+2] ^ mul2[plainText[(4*i)+3]]);
    }

    for (int i = 0; i < 16; ++i)
    {
        plainText[i] = tempC[i];
    }

}
void inverseMixedColumn (unsigned char * plainText)
{
    unsigned char tempC [18];

    for (int i = 0; i < 4; ++i)
    {
        tempC[(4*i)+0] = (unsigned char) (mul_14[plainText[(4*i)+0]] ^ mul_11[plainText[(4*i)+1]] ^ mul_13[plainText[(4*i)+2]] ^ mul_9[plainText[(4*i)+3]]);
        tempC[(4*i)+1] = (unsigned char) (mul_9[plainText[(4*i)+0]] ^ mul_14[plainText[(4*i)+1]] ^ mul_11[plainText[(4*i)+2]] ^ mul_13[plainText[(4*i)+3]]);
        tempC[(4*i)+2] = (unsigned char) (mul_13[plainText[(4*i)+0]] ^ mul_9[plainText[(4*i)+1]] ^ mul_14[plainText[(4*i)+2]] ^ mul_11[plainText[(4*i)+3]]);
        tempC[(4*i)+3] = (unsigned char) (mul_11[plainText[(4*i)+0]] ^ mul_13[plainText[(4*i)+1]] ^ mul_9[plainText[(4*i)+2]] ^ mul_14[plainText[(4*i)+3]]);
    }
    for (int i = 0; i < 16; ++i)
    {
        plainText[i] = tempC[i];
    }

}
void byteSubShiftRow(unsigned char * state)
{

    unsigned char tmp[16];

    tmp[0] = s[state[0]];
    tmp[1] = s[state[5]];
    tmp[2] = s[state[10]];
    tmp[3] = s[state[15]];

    tmp[4] = s[state[4]];
    tmp[5] = s[state[9]];
    tmp[6] = s[state[14]];
    tmp[7] = s[state[3]];

    tmp[8] = s[state[8]];
    tmp[9] = s[state[13]];
    tmp[10] = s[state[2]];
    tmp[11] = s[state[7]];

    tmp[12] = s[state[12]];
    tmp[13] = s[state[1]];
    tmp[14] = s[state[6]];
    tmp[15] = s[state[11]];


    for(int i=0;i<16;i++)
    {
        state[i] = tmp[i];
    }
}
void inverseByteSubShiftRow(unsigned char * plainText)
{
    unsigned char temp[16];
    temp[0] = inv_s[plainText[0]];
    temp[1] = inv_s[plainText[13]];
    temp[2] = inv_s[plainText[10]];
    temp[3] = inv_s[plainText[7]];
    temp[4] = inv_s[plainText[4]];
    temp[5] = inv_s[plainText[1]];
    temp[6] = inv_s[plainText[14]];
    temp[7] = inv_s[plainText[11]];
    temp[8] = inv_s[plainText[8]];
    temp[9] = inv_s[plainText[5]];
    temp[10] = inv_s[plainText[2]];
    temp[11] = inv_s[plainText[15]];
    temp[12] = inv_s[plainText[12]];
    temp[13] = inv_s[plainText[9]];
    temp[14] = inv_s[plainText[6]];
    temp[15] = inv_s[plainText[3]];

    for (int i = 0; i < 16; ++i)
        plainText[i] = temp[i];

}


void AESEncryption(unsigned char * plainText, unsigned char * expandedKey, unsigned char * cipher,int len)
{
    for(int pos=0;pos<len;pos+=16){
        unsigned char state [16];
        //unsigned char * expandedKey = malloc(176);
        //expandedKey = keyExpansion(Key);
        //key addition for the first round
        for (int i = 0; i < 16; ++i)
        {
         state[i] = plainText[pos+i] ^ expandedKey[i];
        }

        //now the 9 rounds begin
        for(int rounds = 1; rounds<10; rounds++)
        {
            byteSubShiftRow(state);
            mixColumns(state);
            int counter = 0;
            int loc = rounds*16;
            while(counter<16)
            {
                state[counter] ^= expandedKey[loc];
                loc++;
                counter++;
            }
        }

        //10th round
        byteSubShiftRow(state);
        for(int i=0; i<16;i++)
        {
            cipher[pos+i] = state[i] ^ expandedKey[i+160];
        }
    }

    

}

void AESDecryption(unsigned char * cipher, unsigned char * expandedKey, unsigned char * plainText)
{
    unsigned char  state[16];
    //key whitening
    for (int i = 0; i < 16; ++i)
        state[i] = cipher[i] ^ expandedKey[160+i];

    // 9 rounds of decryption
    for (int rounds = 9; rounds >0 ; rounds--)
    {
        inverseByteSubShiftRow(state);
        int counter = 0;
        int loc = 16*rounds;
        while(counter<16)
        {
            state[counter] ^= expandedKey[loc];
            loc++;
            counter++;
        }
        inverseMixedColumn(state);
    }

    //final 10th round of decryption
    inverseByteSubShiftRow(state);
    for(int i =0; i<16; i++)
        plainText[i] = state[i] ^ expandedKey[i];

}

int main(){
    //the current code is for 16 byte plaintext and 16 byte key, the code will be further improved upon by adding support for 16*n byte plaintexts as well.

for(int max=130; max<140000000; max*=10){



    char *mainText="this aint a game";
    
    char plaintext[max];
    for(int i=0;i<max;i++){
        plaintext[i]=mainText[i%16];
    }
    char *key="2b7e151628aed2a6";
    unsigned char *expandedkey=keyExpansion((unsigned char*)key);
    int len=strlen(plaintext);
    unsigned char cipher[len];
    unsigned char plain[len];
    printf("\nSize of PlainText is : %d\n", len);
    // printf("expanded key is %s\n",expandedkey);
    // exit(0);
    const clock_t begin_time = clock();
    AESEncryption((unsigned char*)plaintext,expandedkey,cipher,len);
    // int ha=0;
    // int j=0,k=0;
    // for(j=0;j<6000;j++){
    //     for(k=0;k<6000;k++){
    //         ha++;
    //     }
    // }
	float runTime = (float)( clock() - begin_time ) /  1000;
	printf("Time for CPU: %fms\n", runTime);
    printf("cipher is %s\n",cipher);
    AESDecryption(cipher, expandedkey,  plain);
    
    unsigned char GPU_Decrypted_plain[len];
    // printf("plain is %s;\n",plain);

    // clock_t t; 
    // t = clock(); 
    // fun(); 
    // t = clock() - t; 
    // double time_taken = ((double)t)/CLOCKS_PER_SEC; // in seconds 
  
    // printf("fun() took %f seconds to execute \n", time_taken); 

    // printf("%s\n%s\n%s",plaintext, key, (char*)expandedkey);
    

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //GPU BLOCK
    {
        

        hipEvent_t start, stop,kernel1,kernel2;
        float time,timeKernel;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        unsigned char *d_s, *d_mul2, *d_mul_3, *d_mul_9, *d_mul_11, *d_mul_13, *d_mul_14, *d_inv_s;
        unsigned char *d_plainText, *d_expandedKey, *d_cipher;

        hipMalloc((void **) &d_s,      256*sizeof(unsigned char));
        hipMalloc((void **) &d_mul2,   256*sizeof(unsigned char));
        hipMalloc((void **) &d_mul_3,  256*sizeof(unsigned char));
        hipMalloc((void **) &d_mul_9,  256*sizeof(unsigned char));
        hipMalloc((void **) &d_mul_11, 256*sizeof(unsigned char));
        hipMalloc((void **) &d_mul_13, 256*sizeof(unsigned char));
        hipMalloc((void **) &d_mul_14, 256*sizeof(unsigned char));
        hipMalloc((void **) &d_inv_s, 256*sizeof(unsigned char));
        
        hipMemcpy(d_s,     s,      256*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(d_mul2,  mul2,   256*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(d_mul_3, mul_3,  256*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(d_mul_9, mul_9,  256*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(d_mul_11,mul_11, 256*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(d_mul_13,mul_13, 256*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(d_mul_14,mul_14, 256*sizeof(unsigned char),hipMemcpyHostToDevice);
        hipMemcpy(d_inv_s, inv_s,  256*sizeof(unsigned char),hipMemcpyHostToDevice);

        hipMalloc((void **) &d_plainText,  len*sizeof(char));
        hipMalloc((void **) &d_expandedKey,176*sizeof(unsigned char));
        hipMalloc((void **) &d_cipher,     len*sizeof(char));
        
        hipMemcpy(d_plainText,     plaintext,  len*sizeof(char),hipMemcpyHostToDevice);
        hipMemcpy(d_expandedKey,   expandedkey,176*sizeof(unsigned char),hipMemcpyHostToDevice);

        dim3 dimGrid(ceil((float) len/256),1,1); 
        dim3 dimBlock(256,1,1);

        hipEventCreate(&kernel1);
        hipEventCreate(&kernel2);
        hipEventRecord(kernel1, 0);

        GPU_AESEncryption<<<dimGrid,dimBlock>>>(d_plainText,d_expandedKey,d_cipher,d_s,d_mul2,d_mul_3,len);  
        GPU_AESDecryption<<<dimGrid,dimBlock>>>(d_plainText,d_expandedKey,d_cipher,d_inv_s,d_mul_14,d_mul_9,d_mul_13,d_mul_11,len);  
            
        hipEventRecord(kernel2, 0);
        hipEventSynchronize(kernel2);
        hipEventElapsedTime(&timeKernel, kernel1, kernel2);

        hipMemcpy(cipher, d_cipher , len*sizeof(char) , hipMemcpyDeviceToHost);
        hipMemcpy(GPU_Decrypted_plain, d_plainText , len*sizeof(char) , hipMemcpyDeviceToHost);

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);

        printf("GPU cipher is %s\n",cipher);
        printf("GPU Decrypted plain is %s\n",GPU_Decrypted_plain);
        printf("Normal  plain text  is %s\n",plaintext);
        break;
        printf("Time for GPU: %fms\n", time);
        printf("Time for Kernel: %fms\n", timeKernel);
        printf("Speed Up Kernel :  %f\n",runTime/timeKernel);
        printf("Speed Up Total  :  %f\n\n",runTime/time);


        hipFree(d_s);
        hipFree(d_mul2);
        hipFree(d_mul_3);
        hipFree(d_mul_9);
        hipFree(d_mul_11);
        hipFree(d_mul_13);
        hipFree(d_mul_14);

        hipFree(plaintext);
        hipFree(expandedkey);
        hipFree(cipher);

    }
}
    return 0;

}